#include "hip/hip_runtime.h"

struct Individual {
  double fitness;
}

__global__ void evolve(Individual* pop, Individual* boat)
{
	__shared__ Individual oldPop[POP_PER_ISLAND];
	__shared__ Individual newPop[POP_PER_ISLAND];
	int g;
	
	oldPop[threadIdx.x] = pop[threadIdx.x + blockIdx.x * blockDim.x];
	boat[blockIdx.x] = oldPop[0];
	
	for(g = 0; g < 100; g++)
	{
		fitness(oldPop[threadIdx.x]);
		
		int a = rand() % POP_PER_ISLAND;
		int b = rand() % POP_PER_ISLAND;
		newPop[threadIdx.x] = evaluation(oldPop[a], oldPop[b]);
		
		mutate(newPop[threadIdx.x]);
		crossover(newPop[threadIdx.x]);

		boat[(bockIdx.x + 1) % NUM_ISLANDS] = newPop[0];
		__syncthreads()
		newPop[0] = boat[blockIdx.x];
		
		__syncthreads();
		
		oldPop[threadIdx.x] = newPop[threadIdx.x];
	}
}

__device__ Individual evaluation(Individual a, Individual b)
{
	if(a.fitness > b.fitness)
	{
		return a;
	}
	else
	{
		return b;
	}
}
