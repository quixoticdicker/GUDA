#include "hip/hip_runtime.h"
#define NUM_ISLANDS 8
#define POP_PER_ISLAND 256

__global__ void evolve(Individual* pop, Individual* boat)
{
	__shared__ Individual oldPop[POP_PER_ISLAND];
	__shared__ Individual newPop[POP_PER_ISLAND];
	int g;
	
	oldPop[threadIdx.x] = pop[threadIdx.x + blockIdx.x * blockDim.x];
	boat[blockIdx.x] = oldPop[0];
	
	for(g = 0; g < 100; g++)
	{
		fitness(oldPop[threadIdx.x]);
		
		int a = rand() % POP_PER_ISLAND;
		int b = rand() % POP_PER_ISLAND;
		newPop[threadIdx.x] = evaluation(oldPop[a], oldPop[b]);
		
		mutate(newPop[threadIdx.x]);
		crossover(newPop[threadIdx.x]);

		boat[(bockIdx.x + 1) % NUM_ISLANDS] = newPop[0];
		__syncthreads()
		newPop[0] = boat[blockIdx.x];
		
		__syncthreads();
		
		oldPop[threadIdx.x] = newPop[threadIdx.x];
	}

	pop[threadIdx.x + blockIdx.x * blockDim.x] = newPop[threadIdx.x];
}

__device__ Individual evaluation(Individual a, Individual b)
{
	if(a.fitness > b.fitness)
	{
		return a;
	}
	else
	{
		return b;
	}
}

int main()
{
    Individual pop[NUM_ISLANDS * POP_PER_ISLAND];
    Individual *d_pop;
    hipMalloc((void**) &d_pop,
	       sizeof(Individual) * (NUM_ISLANDS * POP_PER_ISLAND));
    hipMemcpy(d_pop, pop, sizeof(Individual) * (NUM_ISLANDS * POP_PER_ISLAND),
	hipMemcpyHostToDevice);

    Individual boat[NUM_ISLANDS];
    Individual *d_boat;
    hipMalloc((void**) &d_boat, sizeof(Individual) * NUM_ISLANDS);
    hipMemcpy(d_boat, boat, sizeof(Individual) * NUM_ISLANDS);

    evolve<<<NUM_ISLANDS, POP_PER_ISLAND>>>(d_pop, d_boat);

    hipMemcpy(pop, d_pop, sizeof(Individual) * (NUM_ISLANDS * POP_PER_ISLAND),
	hipMemcpyDeviceToHost);

}
